

#include <query_compilation/gpu_handbuilt/queries.h>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>
#include <boost/make_shared.hpp>
#include <core/column.hpp>
#include <core/global_definitions.hpp>
#include <core/memory_allocator.hpp>
#include <iostream>
#include <limits>
#include <persistence/storage_manager.hpp>
#include <vector>
#include "core/variable_manager.hpp"
// for dictionary compressed column
#include <compression/dictionary_compressed_column.hpp>
#include <util/getname.hpp>
// hashtable
#include <query_compilation/gpu_utilities/util/cuda_griddim.h>
#include <query_compilation/gpu_utilities/util/divup.h>
#include <query_compilation/gpu_utilities/compilation_hashtables/GPU_CHT32_2.cuh>
// reduce (sort, limit)
#include <algorithm>
#include <query_compilation/gpu_utilities/util/sequence.cuh>

#include "tpch7.cuh"

// -- TPC-H Query 7
// select
//         supp_nation,
//         cust_nation,
//         l_year,
//         sum(volume) as revenue
// from
//         (
//                 select
//                         n1.n_name as supp_nation,
//                         n2.n_name as cust_nation,
//                         extract(year from l_shipdate) as l_year,
//                         l_extendedprice * (1 - l_discount) as volume
//                 from
//                         supplier,
//                         lineitem,
//                         orders,
//                         customer,
//                         nation n1,
//                         nation n2
//                 where
//                         s_suppkey = l_suppkey
//                         and o_orderkey = l_orderkey
//                         and c_custkey = o_custkey
//                         and s_nationkey = n1.n_nationkey
//                         and c_nationkey = n2.n_nationkey
//                         and (
//                                 (n1.n_name = 'FRANCE' and n2.n_name =
//                                 'GERMANY')
//                                 or (n1.n_name = 'GERMANY' and n2.n_name =
//                                 'FRANCE')
//                         )
//                         and l_shipdate between date '1995-01-01' and date
//                         '1996-12-31'
//         ) as shipping
// group by
//         supp_nation,
//         cust_nation,
//         l_year
// order by
//         supp_nation,
//         cust_nation,
//         l_year

using namespace std;

namespace CoGaDB {

TablePtr tpch7();

bool tpch7_holistic_kernel(ClientPtr client) {
  tpch7();
  return true;
}

TablePtr tpch7() {
  CUDA_CHECK_ERROR_RETURN("Initial failure");

  // build nation ht

  // build orders ht

  // build customer ht

  // build suppliers ht

  // filter shipdate

  // probe lineitem -> supplier -> nation

  // probe lineitem -> orders -> customer -> nation

  // check nation

  // aggregate sum(revenue) by nation x nation x year

  int numGroups = 25;
  int numProbeBlocks = divUp(lineitem_num_elements, (size_t)kProbeBlockSize);
  float *d_groupAggregatePerBlock =
      customMalloc<float>(DEVICE_MEMORY, numProbeBlocks * numGroups);
  int *d_groupCountPerBlock =
      customMalloc<int>(DEVICE_MEMORY, numProbeBlocks * numGroups);
  vector<float> groupAggregatePerBlock(numProbeBlocks * numGroups);
  vector<int> groupCountPerBlock(numProbeBlocks * numGroups);
  Timestamp begin_query = getTimestamp();

  double region_selectivity = 1;
  GPU_CHT32_2::HashTable regionHashTable = GPU_CHT32_2::createHashTable(
      region_num_elements * region_selectivity, 1.25);
  dim3 gridDim =
      computeGridDim(region_num_elements, kGridSize, kBuildBlockSize);

  regionBuildKernel<<<gridDim, kBuildBlockSize>>>(
      regionHashTable, d_region_regionkey, d_region_name, region_num_elements);

  double nation_selectivity = 1;
  GPU_CHT32_2::HashTable nationHashTable = GPU_CHT32_2::createHashTable(
      nation_num_elements * nation_selectivity, 1.25);
  gridDim = computeGridDim(nation_num_elements, kGridSize, kBuildBlockSize);

  nationBuildKernel<<<gridDim, kBuildBlockSize>>>(
      nationHashTable, d_nation_nationkey, nation_num_elements);

  double supplier_selectivity = 1;
  GPU_CHT32_2::HashTable supplierHashTable = GPU_CHT32_2::createHashTable(
      supplier_num_elements * supplier_selectivity, 1.25);
  gridDim = computeGridDim(supplier_num_elements, kGridSize, kBuildBlockSize);

  supplierBuildKernel<<<gridDim, kBuildBlockSize>>>(
      supplierHashTable, d_supplier_suppkey, supplier_num_elements);

  double customer_selectivity = 1;
  GPU_CHT32_2::HashTable customerHashTable = GPU_CHT32_2::createHashTable(
      customer_num_elements * customer_selectivity, 1.25);
  gridDim = computeGridDim(customer_num_elements, kGridSize, kBuildBlockSize);

  customerBuildKernel<<<gridDim, kBuildBlockSize>>>(
      customerHashTable, d_customer_custkey, customer_num_elements);

  double orders_selectivity = 1;
  GPU_CHT32_2::HashTable ordersHashTable = GPU_CHT32_2::createHashTable(
      orders_num_elements * orders_selectivity, 1.25);
  gridDim = computeGridDim(orders_num_elements, kGridSize, kBuildBlockSize);

  ordersBuildKernel<<<gridDim, kBuildBlockSize>>>(
      ordersHashTable, d_orders_orderkey, d_orders_orderdate,
      orders_num_elements);

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR_RETURN("build failed");

  Timestamp end_build_start_probe = getTimestamp();

  gridDim = computeGridDim(lineitem_num_elements, kGridSize, kProbeBlockSize);
  lineitemProbeKernel<<<gridDim, kProbeBlockSize>>>(
      regionHashTable, nationHashTable, supplierHashTable, customerHashTable,
      ordersHashTable, d_lineitem_suppkey, d_lineitem_orderkey,
      d_supplier_nationkey, d_orders_custkey, d_nation_regionkey,
      d_customer_nationkey, lineitem_num_elements, d_lineitem_extended_price,
      d_lineitem_discount, d_nation_name, numGroups, d_groupAggregatePerBlock,
      d_groupCountPerBlock);

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR_RETURN("probe failed");

  Timestamp end_probe_start_reduce = getTimestamp();

  hipMemcpy(&groupAggregatePerBlock[0], d_groupAggregatePerBlock,
             sizeof(float) * numProbeBlocks * numGroups,
             hipMemcpyDeviceToHost);
  hipMemcpy(&groupCountPerBlock[0], d_groupCountPerBlock,
             sizeof(int) * numProbeBlocks * numGroups, hipMemcpyDeviceToHost);
  CUDA_CHECK_ERROR_RETURN("probe failed");
  vector<float> aggregateResult(numGroups);
  vector<int> aggregateCount(numGroups);
  for (int i = 0; i < numProbeBlocks; i++) {
    for (int j = 0; j < numGroups; j++) {
      aggregateResult[j] += groupAggregatePerBlock[i * numGroups + j];
      aggregateCount[j] += groupCountPerBlock[i * numGroups + j];
    }
  }

  Timestamp end_query = getTimestamp();

  GPU_CHT32_2::printHashTable(regionHashTable);
  GPU_CHT32_2::printHashTable(nationHashTable);
  GPU_CHT32_2::printHashTable(supplierHashTable);
  GPU_CHT32_2::printHashTable(customerHashTable);
  GPU_CHT32_2::printHashTable(ordersHashTable);

  for (int j = 0; j < numGroups; j++) {
    if (aggregateResult[j] > 0.0)
      cout << nation_name->reverseLookup(j) << ", " << aggregateResult[j]
           << ", " << aggregateCount[j] << endl;
  }

  double build_time =
      double(end_build_start_probe - begin_query) / (1000 * 1000);
  double probe_time =
      double(end_probe_start_reduce - end_build_start_probe) / (1000 * 1000);
  double reduce_time =
      double(end_query - end_probe_start_reduce) / (1000 * 1000);
  double query_time = double(end_query - begin_query) / (1000 * 1000);

  cout << "TPCH5 execution time " << query_time << " ms" << endl;
  cout << "build: " << build_time << " ms, "
       << "probe: " << probe_time << " ms, "
       << "reduce: " << reduce_time << " ms, " << endl;

  GPU_CHT32_2::freeHashTable(regionHashTable);
  GPU_CHT32_2::freeHashTable(nationHashTable);
  GPU_CHT32_2::freeHashTable(supplierHashTable);
  GPU_CHT32_2::freeHashTable(customerHashTable);
  GPU_CHT32_2::freeHashTable(ordersHashTable);

  customFree<float>(DEVICE_MEMORY, d_groupAggregatePerBlock);
  customFree<int>(DEVICE_MEMORY, d_groupCountPerBlock);

  customFree<int>(DEVICE_MEMORY, d_lineitem_orderkey);
  customFree<int>(DEVICE_MEMORY, d_lineitem_suppkey);
  customFree<float>(DEVICE_MEMORY, d_lineitem_discount);
  customFree<float>(DEVICE_MEMORY, d_lineitem_extended_price);
  customFree<int>(DEVICE_MEMORY, d_customer_custkey);
  customFree<int>(DEVICE_MEMORY, d_customer_nationkey);
  customFree<uint32_t>(DEVICE_MEMORY, d_orders_orderdate);
  customFree<int>(DEVICE_MEMORY, d_orders_custkey);
  customFree<int>(DEVICE_MEMORY, d_orders_orderkey);
  customFree<int>(DEVICE_MEMORY, d_supplier_suppkey);
  customFree<int>(DEVICE_MEMORY, d_supplier_nationkey);
  customFree<int>(DEVICE_MEMORY, d_nation_nationkey);
  customFree<uint32_t>(DEVICE_MEMORY, d_nation_name);
  customFree<int>(DEVICE_MEMORY, d_nation_regionkey);
  customFree<int>(DEVICE_MEMORY, d_region_regionkey);
  customFree<uint32_t>(DEVICE_MEMORY, d_region_name);

  return TablePtr();
}
}
