

#include <query_compilation/gpu_handbuilt/queries.h>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>
#include <boost/make_shared.hpp>
#include <core/column.hpp>
#include <core/global_definitions.hpp>
#include <core/memory_allocator.hpp>
#include <iostream>
#include <limits>
#include <persistence/storage_manager.hpp>
#include <vector>
#include "core/variable_manager.hpp"
// for dictionary compressed column
#include <compression/dictionary_compressed_column.hpp>
#include <util/getname.hpp>
// hashtable
#include <query_compilation/gpu_utilities/util/cuda_griddim.h>
#include <query_compilation/gpu_utilities/util/divup.h>
#include <query_compilation/gpu_utilities/compilation_hashtables/GPU_CHT32_2.cuh>
// reduce (sort, limit)
#include <algorithm>
#include <query_compilation/gpu_utilities/util/sequence.cuh>

#include "tpch10.cuh"

// -- TPC-H Query 1
// select
//         l_returnflag,
//         l_linestatus,
//         sum(l_quantity) as sum_qty,
//         sum(l_extendedprice) as sum_base_price,
//         sum(l_extendedprice * (1 - l_discount)) as sum_disc_price,
//         sum(l_extendedprice * (1 - l_discount) * (1 + l_tax)) as sum_charge,
//         avg(l_quantity) as avg_qty,
//         avg(l_extendedprice) as avg_price,
//         avg(l_discount) as avg_disc,
//         count(*) as count_order
// from
//         lineitem
// where
//         l_shipdate <= date '1998-12-01' - interval '90' day
// group by
//         l_returnflag,
//         l_linestatus
// order by
//         l_returnflag,
//         l_linestatus

using namespace std;

namespace CoGaDB {

TablePtr tpch1();

bool tpch1_holistic_kernel(ClientPtr client) {
  tpch1();
  return true;
}

TablePtr tpch1() {
  CUDA_CHECK_ERROR_RETURN("Initial failure");

  // scan lineitem aggregate many attributes group by returnflag x linestatus

  int numGroups = 25;
  int numProbeBlocks = divUp(lineitem_num_elements, (size_t)kProbeBlockSize);
  float *d_groupAggregatePerBlock =
      customMalloc<float>(DEVICE_MEMORY, numProbeBlocks * numGroups);
  int *d_groupCountPerBlock =
      customMalloc<int>(DEVICE_MEMORY, numProbeBlocks * numGroups);
  vector<float> groupAggregatePerBlock(numProbeBlocks * numGroups);
  vector<int> groupCountPerBlock(numProbeBlocks * numGroups);
  Timestamp begin_query = getTimestamp();

  double region_selectivity = 1;
  GPU_CHT32_2::HashTable regionHashTable = GPU_CHT32_2::createHashTable(
      region_num_elements * region_selectivity, 1.25);
  dim3 gridDim =
      computeGridDim(region_num_elements, kGridSize, kBuildBlockSize);

  regionBuildKernel<<<gridDim, kBuildBlockSize>>>(
      regionHashTable, d_region_regionkey, d_region_name, region_num_elements);

  double nation_selectivity = 1;
  GPU_CHT32_2::HashTable nationHashTable = GPU_CHT32_2::createHashTable(
      nation_num_elements * nation_selectivity, 1.25);
  gridDim = computeGridDim(nation_num_elements, kGridSize, kBuildBlockSize);

  nationBuildKernel<<<gridDim, kBuildBlockSize>>>(
      nationHashTable, d_nation_nationkey, nation_num_elements);

  double supplier_selectivity = 1;
  GPU_CHT32_2::HashTable supplierHashTable = GPU_CHT32_2::createHashTable(
      supplier_num_elements * supplier_selectivity, 1.25);
  gridDim = computeGridDim(supplier_num_elements, kGridSize, kBuildBlockSize);

  supplierBuildKernel<<<gridDim, kBuildBlockSize>>>(
      supplierHashTable, d_supplier_suppkey, supplier_num_elements);

  double customer_selectivity = 1;
  GPU_CHT32_2::HashTable customerHashTable = GPU_CHT32_2::createHashTable(
      customer_num_elements * customer_selectivity, 1.25);
  gridDim = computeGridDim(customer_num_elements, kGridSize, kBuildBlockSize);

  customerBuildKernel<<<gridDim, kBuildBlockSize>>>(
      customerHashTable, d_customer_custkey, customer_num_elements);

  double orders_selectivity = 1;
  GPU_CHT32_2::HashTable ordersHashTable = GPU_CHT32_2::createHashTable(
      orders_num_elements * orders_selectivity, 1.25);
  gridDim = computeGridDim(orders_num_elements, kGridSize, kBuildBlockSize);

  ordersBuildKernel<<<gridDim, kBuildBlockSize>>>(
      ordersHashTable, d_orders_orderkey, d_orders_orderdate,
      orders_num_elements);

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR_RETURN("build failed");

  Timestamp end_build_start_probe = getTimestamp();

  gridDim = computeGridDim(lineitem_num_elements, kGridSize, kProbeBlockSize);
  lineitemProbeKernel<<<gridDim, kProbeBlockSize>>>(
      regionHashTable, nationHashTable, supplierHashTable, customerHashTable,
      ordersHashTable, d_lineitem_suppkey, d_lineitem_orderkey,
      d_supplier_nationkey, d_orders_custkey, d_nation_regionkey,
      d_customer_nationkey, lineitem_num_elements, d_lineitem_extended_price,
      d_lineitem_discount, d_nation_name, numGroups, d_groupAggregatePerBlock,
      d_groupCountPerBlock);

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR_RETURN("probe failed");

  Timestamp end_probe_start_reduce = getTimestamp();

  hipMemcpy(&groupAggregatePerBlock[0], d_groupAggregatePerBlock,
             sizeof(float) * numProbeBlocks * numGroups,
             hipMemcpyDeviceToHost);
  hipMemcpy(&groupCountPerBlock[0], d_groupCountPerBlock,
             sizeof(int) * numProbeBlocks * numGroups, hipMemcpyDeviceToHost);
  CUDA_CHECK_ERROR_RETURN("probe failed");
  vector<float> aggregateResult(numGroups);
  vector<int> aggregateCount(numGroups);
  for (int i = 0; i < numProbeBlocks; i++) {
    for (int j = 0; j < numGroups; j++) {
      aggregateResult[j] += groupAggregatePerBlock[i * numGroups + j];
      aggregateCount[j] += groupCountPerBlock[i * numGroups + j];
    }
  }

  Timestamp end_query = getTimestamp();

  GPU_CHT32_2::printHashTable(regionHashTable);
  GPU_CHT32_2::printHashTable(nationHashTable);
  GPU_CHT32_2::printHashTable(supplierHashTable);
  GPU_CHT32_2::printHashTable(customerHashTable);
  GPU_CHT32_2::printHashTable(ordersHashTable);

  for (int j = 0; j < numGroups; j++) {
    if (aggregateResult[j] > 0.0)
      cout << nation_name->reverseLookup(j) << ", " << aggregateResult[j]
           << ", " << aggregateCount[j] << endl;
  }

  double build_time =
      double(end_build_start_probe - begin_query) / (1000 * 1000);
  double probe_time =
      double(end_probe_start_reduce - end_build_start_probe) / (1000 * 1000);
  double reduce_time =
      double(end_query - end_probe_start_reduce) / (1000 * 1000);
  double query_time = double(end_query - begin_query) / (1000 * 1000);

  cout << "TPCH5 execution time " << query_time << " ms" << endl;
  cout << "build: " << build_time << " ms, "
       << "probe: " << probe_time << " ms, "
       << "reduce: " << reduce_time << " ms, " << endl;

  GPU_CHT32_2::freeHashTable(regionHashTable);
  GPU_CHT32_2::freeHashTable(nationHashTable);
  GPU_CHT32_2::freeHashTable(supplierHashTable);
  GPU_CHT32_2::freeHashTable(customerHashTable);
  GPU_CHT32_2::freeHashTable(ordersHashTable);

  customFree<float>(DEVICE_MEMORY, d_groupAggregatePerBlock);
  customFree<int>(DEVICE_MEMORY, d_groupCountPerBlock);

  customFree<int>(DEVICE_MEMORY, d_lineitem_orderkey);
  customFree<int>(DEVICE_MEMORY, d_lineitem_suppkey);
  customFree<float>(DEVICE_MEMORY, d_lineitem_discount);
  customFree<float>(DEVICE_MEMORY, d_lineitem_extended_price);
  customFree<int>(DEVICE_MEMORY, d_customer_custkey);
  customFree<int>(DEVICE_MEMORY, d_customer_nationkey);
  customFree<uint32_t>(DEVICE_MEMORY, d_orders_orderdate);
  customFree<int>(DEVICE_MEMORY, d_orders_custkey);
  customFree<int>(DEVICE_MEMORY, d_orders_orderkey);
  customFree<int>(DEVICE_MEMORY, d_supplier_suppkey);
  customFree<int>(DEVICE_MEMORY, d_supplier_nationkey);
  customFree<int>(DEVICE_MEMORY, d_nation_nationkey);
  customFree<uint32_t>(DEVICE_MEMORY, d_nation_name);
  customFree<int>(DEVICE_MEMORY, d_nation_regionkey);
  customFree<int>(DEVICE_MEMORY, d_region_regionkey);
  customFree<uint32_t>(DEVICE_MEMORY, d_region_name);

  return TablePtr();
}
}
