

#include <query_compilation/gpu_handbuilt/queries.h>
#include <thrust/device_ptr.h>
#include <thrust/transform_reduce.h>
#include <boost/make_shared.hpp>
#include <core/column.hpp>
#include <core/global_definitions.hpp>
#include <core/memory_allocator.hpp>
#include <iostream>
#include <limits>
#include <persistence/storage_manager.hpp>
#include <vector>
#include "core/variable_manager.hpp"
// for dictionary compressed column
#include <compression/dictionary_compressed_column.hpp>
#include <util/getname.hpp>
// hashtable
#include <query_compilation/gpu_utilities/util/cuda_griddim.h>
#include <query_compilation/gpu_utilities/util/divup.h>
#include <query_compilation/gpu_utilities/compilation_hashtables/GPU_CHT32_2.cuh>
// reduce (sort, limit)
#include <algorithm>
#include <query_compilation/gpu_utilities/util/sequence.cuh>

#include "tpch10.cuh"

// -- TPC-H Query 10
// select
//         c_custkey,
//         c_name,
//         sum(l_extendedprice * (1 - l_discount)) as revenue,
//         c_acctbal,
//         n_name,
//         c_address,
//         c_phone,
//         c_comment
// from
//         customer,
//         orders,
//         lineitem,
//         nation
// where
//         c_custkey = o_custkey
//         and l_orderkey = o_orderkey
//         and o_orderdate >= date '1993-10-01'
//         and o_orderdate < date '1994-01-01'
//         and l_returnflag = 'R'
//         and c_nationkey = n_nationkey
// group by
//         c_custkey,
//         c_name,
//         c_acctbal,
//         c_phone,
//         n_name,
//         c_address,
//         c_comment
// order by
//         revenue desc
// limit 20

using namespace std;

namespace CoGaDB {

TablePtr tpch10();

bool tpch10_holistic_kernel(ClientPtr client) {
  tpch10();
  return true;
}

TablePtr tpch10() {
  CUDA_CHECK_ERROR_RETURN("Initial failure");

  // build customer ht

  // build orders ht (filter)

  // build nation ht

  // probe (filter) lineitem -> orders -> customer -> nation
  // aggregate revenue group by customer

  int numGroups = 25;
  int numProbeBlocks = divUp(lineitem_num_elements, (size_t)kProbeBlockSize);
  float *d_groupAggregatePerBlock =
      customMalloc<float>(DEVICE_MEMORY, numProbeBlocks * numGroups);
  int *d_groupCountPerBlock =
      customMalloc<int>(DEVICE_MEMORY, numProbeBlocks * numGroups);
  vector<float> groupAggregatePerBlock(numProbeBlocks * numGroups);
  vector<int> groupCountPerBlock(numProbeBlocks * numGroups);
  Timestamp begin_query = getTimestamp();

  double region_selectivity = 1;
  GPU_CHT32_2::HashTable regionHashTable = GPU_CHT32_2::createHashTable(
      region_num_elements * region_selectivity, 1.25);
  dim3 gridDim =
      computeGridDim(region_num_elements, kGridSize, kBuildBlockSize);

  regionBuildKernel<<<gridDim, kBuildBlockSize>>>(
      regionHashTable, d_region_regionkey, d_region_name, region_num_elements);

  double nation_selectivity = 1;
  GPU_CHT32_2::HashTable nationHashTable = GPU_CHT32_2::createHashTable(
      nation_num_elements * nation_selectivity, 1.25);
  gridDim = computeGridDim(nation_num_elements, kGridSize, kBuildBlockSize);

  nationBuildKernel<<<gridDim, kBuildBlockSize>>>(
      nationHashTable, d_nation_nationkey, nation_num_elements);

  double supplier_selectivity = 1;
  GPU_CHT32_2::HashTable supplierHashTable = GPU_CHT32_2::createHashTable(
      supplier_num_elements * supplier_selectivity, 1.25);
  gridDim = computeGridDim(supplier_num_elements, kGridSize, kBuildBlockSize);

  supplierBuildKernel<<<gridDim, kBuildBlockSize>>>(
      supplierHashTable, d_supplier_suppkey, supplier_num_elements);

  double customer_selectivity = 1;
  GPU_CHT32_2::HashTable customerHashTable = GPU_CHT32_2::createHashTable(
      customer_num_elements * customer_selectivity, 1.25);
  gridDim = computeGridDim(customer_num_elements, kGridSize, kBuildBlockSize);

  customerBuildKernel<<<gridDim, kBuildBlockSize>>>(
      customerHashTable, d_customer_custkey, customer_num_elements);

  double orders_selectivity = 1;
  GPU_CHT32_2::HashTable ordersHashTable = GPU_CHT32_2::createHashTable(
      orders_num_elements * orders_selectivity, 1.25);
  gridDim = computeGridDim(orders_num_elements, kGridSize, kBuildBlockSize);

  ordersBuildKernel<<<gridDim, kBuildBlockSize>>>(
      ordersHashTable, d_orders_orderkey, d_orders_orderdate,
      orders_num_elements);

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR_RETURN("build failed");

  Timestamp end_build_start_probe = getTimestamp();

  gridDim = computeGridDim(lineitem_num_elements, kGridSize, kProbeBlockSize);
  lineitemProbeKernel<<<gridDim, kProbeBlockSize>>>(
      regionHashTable, nationHashTable, supplierHashTable, customerHashTable,
      ordersHashTable, d_lineitem_suppkey, d_lineitem_orderkey,
      d_supplier_nationkey, d_orders_custkey, d_nation_regionkey,
      d_customer_nationkey, lineitem_num_elements, d_lineitem_extended_price,
      d_lineitem_discount, d_nation_name, numGroups, d_groupAggregatePerBlock,
      d_groupCountPerBlock);

  hipDeviceSynchronize();
  CUDA_CHECK_ERROR_RETURN("probe failed");

  Timestamp end_probe_start_reduce = getTimestamp();

  hipMemcpy(&groupAggregatePerBlock[0], d_groupAggregatePerBlock,
             sizeof(float) * numProbeBlocks * numGroups,
             hipMemcpyDeviceToHost);
  hipMemcpy(&groupCountPerBlock[0], d_groupCountPerBlock,
             sizeof(int) * numProbeBlocks * numGroups, hipMemcpyDeviceToHost);
  CUDA_CHECK_ERROR_RETURN("probe failed");
  vector<float> aggregateResult(numGroups);
  vector<int> aggregateCount(numGroups);
  for (int i = 0; i < numProbeBlocks; i++) {
    for (int j = 0; j < numGroups; j++) {
      aggregateResult[j] += groupAggregatePerBlock[i * numGroups + j];
      aggregateCount[j] += groupCountPerBlock[i * numGroups + j];
    }
  }

  Timestamp end_query = getTimestamp();

  GPU_CHT32_2::printHashTable(regionHashTable);
  GPU_CHT32_2::printHashTable(nationHashTable);
  GPU_CHT32_2::printHashTable(supplierHashTable);
  GPU_CHT32_2::printHashTable(customerHashTable);
  GPU_CHT32_2::printHashTable(ordersHashTable);

  for (int j = 0; j < numGroups; j++) {
    if (aggregateResult[j] > 0.0)
      cout << nation_name->reverseLookup(j) << ", " << aggregateResult[j]
           << ", " << aggregateCount[j] << endl;
  }

  double build_time =
      double(end_build_start_probe - begin_query) / (1000 * 1000);
  double probe_time =
      double(end_probe_start_reduce - end_build_start_probe) / (1000 * 1000);
  double reduce_time =
      double(end_query - end_probe_start_reduce) / (1000 * 1000);
  double query_time = double(end_query - begin_query) / (1000 * 1000);

  cout << "TPCH5 execution time " << query_time << " ms" << endl;
  cout << "build: " << build_time << " ms, "
       << "probe: " << probe_time << " ms, "
       << "reduce: " << reduce_time << " ms, " << endl;

  GPU_CHT32_2::freeHashTable(regionHashTable);
  GPU_CHT32_2::freeHashTable(nationHashTable);
  GPU_CHT32_2::freeHashTable(supplierHashTable);
  GPU_CHT32_2::freeHashTable(customerHashTable);
  GPU_CHT32_2::freeHashTable(ordersHashTable);

  customFree<float>(DEVICE_MEMORY, d_groupAggregatePerBlock);
  customFree<int>(DEVICE_MEMORY, d_groupCountPerBlock);

  customFree<int>(DEVICE_MEMORY, d_lineitem_orderkey);
  customFree<int>(DEVICE_MEMORY, d_lineitem_suppkey);
  customFree<float>(DEVICE_MEMORY, d_lineitem_discount);
  customFree<float>(DEVICE_MEMORY, d_lineitem_extended_price);
  customFree<int>(DEVICE_MEMORY, d_customer_custkey);
  customFree<int>(DEVICE_MEMORY, d_customer_nationkey);
  customFree<uint32_t>(DEVICE_MEMORY, d_orders_orderdate);
  customFree<int>(DEVICE_MEMORY, d_orders_custkey);
  customFree<int>(DEVICE_MEMORY, d_orders_orderkey);
  customFree<int>(DEVICE_MEMORY, d_supplier_suppkey);
  customFree<int>(DEVICE_MEMORY, d_supplier_nationkey);
  customFree<int>(DEVICE_MEMORY, d_nation_nationkey);
  customFree<uint32_t>(DEVICE_MEMORY, d_nation_name);
  customFree<int>(DEVICE_MEMORY, d_nation_regionkey);
  customFree<int>(DEVICE_MEMORY, d_region_regionkey);
  customFree<uint32_t>(DEVICE_MEMORY, d_region_name);

  return TablePtr();
}
}
